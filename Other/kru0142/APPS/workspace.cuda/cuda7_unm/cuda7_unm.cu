#include "hip/hip_runtime.h"

// ********************************************************************* 
// 
// Demo program for education in subject 
// Computer Architectures and Parallel Systems. 
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava 
// email:petr.olivka@vsb.cz 
// 
// Example of CUDA Technology Usage with unified memory. 
// 
// Image transformation from RGB to BW schema.  
// 
// ********************************************************************* 
 
#include <stdio.h> 
#include <cuda_device_runtime_api.h> 
#include <hip/hip_runtime.h> 
#include <math.h> 
#include "pic_type.h" 
 
// Demo kernel to transform RGB color schema to BW schema 
 
__global__ void kernel_zmenseni( CUDA_Pic colorPic, CUDA_Pic bwPic ) 
{ 
    // X,Y coordinates and check image dimensions 
    int y = blockDim.y * blockIdx.y + threadIdx.y; 
    int x = blockDim.x * blockIdx.x + threadIdx.x; 
    if ( y >= colorPic.Size.y ) return; 
    if ( x >= colorPic.Size.x ) return; 
 
 
    uchar3 bgr[4]; 
    bgr[0] = colorPic.P_uchar3[y * colorPic.Size.x + x]; 
    bgr[1] = colorPic.P_uchar3[y * colorPic.Size.x + x + 1]; 
    bgr[2] = colorPic.P_uchar3[(y + 1) * colorPic.Size.x + x]; 
    bgr[3] = colorPic.P_uchar3[(y + 1) * colorPic.Size.x + x + 1]; 
 
    uchar3 bgr_small; 
    bgr_small.x = (bgr[0].x + bgr[1].x + bgr[2].x + bgr[3].x) / 4; 
    bgr_small.y = (bgr[0].y + bgr[1].y + bgr[2].y + bgr[3].y) / 4; 
    bgr_small.z = (bgr[0].z + bgr[1].z + bgr[2].z + bgr[3].z) / 4; 
    bwPic.P_uchar3[(y/2) * bwPic.Size.x + (x/2)] = bgr_small; 
 
} 
 
 
 
 
__global__ void kernel_rotace( CUDA_Pic colorPic, CUDA_Pic bwPic,  float degr ) 
{ 
     int y = blockDim.y * blockIdx.y + threadIdx.y; 
    int x = blockDim.x * blockIdx.x + threadIdx.x; 
    if ( y >= colorPic.Size.y ) return; 
    if ( x >= colorPic.Size.x ) return; 
 
 
 
    uchar3 bgr = colorPic.P_uchar3[ y * colorPic.Size.x + x ]; 
    float deg = degr*3.14/180; 
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y; 
 
    int xc = colorPic.Size.x - colorPic.Size.x/2; 
    int yc = colorPic.Size.y - colorPic.Size.y/2; 
 
    int newx = ((float)i-xc)*cos(deg) - ((float)j-yc)*sin(deg) + xc*2; 
    int newy = ((float)i-xc)*sin(deg) + ((float)j-yc)*cos(deg) + yc*2; 
    if (newx >= 0 && newx < bwPic.Size.x && newy >= 0 && newy < bwPic.Size.y) 
    { 
        bwPic.P_uchar3[newy*bwPic.Size.x+newx]=bgr; 
    } 
} 
 
 
__global__ void kernel_zvetseni1( CUDA_Pic colorPic, CUDA_Pic bwPic ) 
{ 
    int y = blockDim.y * blockIdx.y + threadIdx.y; 
    int x = blockDim.x * blockIdx.x + threadIdx.x; 
    if ( y >= colorPic.Size.y ) return; 
    if ( x >= colorPic.Size.x ) return; 
    if ( y < 0) return; 
    if ( x < 0 ) return; 
 
    int x1 = x, y1=y; 
 
    bwPic.P_uchar3[y * colorPic.Size.x + x] = colorPic.P_uchar3[y * colorPic.Size.x + x]; 
 
    //bwPic.P_uchar3[y * bwPic.Size.x + x] = colorPic.P_uchar3[y * colorPic.Size.x + x]; 
    /*bwPic.P_uchar3[(y1) * bwPic.Size.x + (x1+1)] = colorPic.P_uchar3[y * colorPic.Size.x + x]; 
    bwPic.P_uchar3[(y1+1) * bwPic.Size.x + (x1)] = colorPic.P_uchar3[y * colorPic.Size.x + x]; 
    bwPic.P_uchar3[(y1+1) * bwPic.Size.x + (x1+1)] = colorPic.P_uchar3[y * colorPic.Size.x + x];*/ 
 
 
} 
 
 
__global__ void kernel_zvetseni2( CUDA_Pic colorPic, CUDA_Pic bwPic ) 
{ 
    int y = blockDim.y * blockIdx.y + threadIdx.y; 
    int x = blockDim.x * blockIdx.x + threadIdx.x; 
    if ( y >= colorPic.Size.y ) return; 
    if ( x >= colorPic.Size.x ) return; 
 
 
    uchar3 bgr[4]; 
    bgr[0] = colorPic.P_uchar3[y * colorPic.Size.x + x]; 
    bgr[1] = colorPic.P_uchar3[y * colorPic.Size.x + x + 1]; 
    bgr[2] = colorPic.P_uchar3[(y + 1) * colorPic.Size.x + x]; 
    bgr[3] = colorPic.P_uchar3[(y + 1) * colorPic.Size.x + x + 1]; 
 
    uchar3 p1, p2, p3, p4; 
 
    p1.x = bgr[0].x; 
    p1.y = bgr[0].y; 
    p1.z = bgr[0].z; 
 
    p2.x = (bgr[0].x + bgr[1].x )/2; 
    p2.y = (bgr[0].y + bgr[1].y )/2; 
    p2.z = (bgr[0].z + bgr[1].z )/2; 
 
    p3.x = (bgr[0].x + bgr[2].x )/2; 
    p3.y = (bgr[0].y + bgr[2].y )/2; 
    p3.z = (bgr[0].z + bgr[2].z )/2; 
 
    p4.x = (bgr[0].x + bgr[3].x )/2; 
    p4.y = (bgr[0].y + bgr[3].y )/2; 
    p4.z = (bgr[0].z + bgr[3].z )/2; 
 
    int x1 = 2*x, y1=2*y; 
 
    bwPic.P_uchar3[(y1) * bwPic.Size.x + (x1)] = p1; 
    bwPic.P_uchar3[(y1) * bwPic.Size.x + (x1+1)] = p2; 
    bwPic.P_uchar3[(y1+1) * bwPic.Size.x + (x1)] = p3; 
    bwPic.P_uchar3[(y1+1) * bwPic.Size.x + (x1+1)] = p4; 
} 
 
__global__ void kernel_zacerneni( CUDA_Pic colorPic) 
{ 
    int y = blockDim.y * blockIdx.y + threadIdx.y; 
    int x = blockDim.x * blockIdx.x + threadIdx.x; 
    if ( y >= colorPic.Size.y ) return; 
    if ( x >= colorPic.Size.x ) return; 
 
    uchar3 bgr = colorPic.P_uchar3[ y * colorPic.Size.x + x ]; 
 
    bgr.x = 0; 
    bgr.y = 0; 
    bgr.z = 0; 
 
    colorPic.P_uchar3[ y * colorPic.Size.x + x ] = bgr; 
 
} 
 
 
void cu_run_zmenseni( CUDA_Pic colorPic, CUDA_Pic bwPic) 
{ 
    hipError_t cerr; 
 
    int block_size = 16; 
    dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size ); 
    dim3 threads( block_size, block_size ); 
    kernel_zmenseni<<< blocks, threads >>>( colorPic, bwPic); 
 
    if ( ( cerr = hipGetLastError() ) != hipSuccess ) 
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) ); 
 
    hipDeviceSynchronize(); 
} 
 
void cu_run_rotace( CUDA_Pic colorPic, CUDA_Pic bwPic, float degr ) 
{ 
    hipError_t cerr; 
 
    int block_size = 16; 
    dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size ); 
    dim3 threads( block_size, block_size ); 
    kernel_rotace<<< blocks, threads >>>( colorPic, bwPic, degr ); 
 
    if ( ( cerr = hipGetLastError() ) != hipSuccess ) 
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) ); 
 
    hipDeviceSynchronize(); 
} 
 
void cu_run_zvetseni1( CUDA_Pic colorPic, CUDA_Pic bwPic ) 
{ 
    hipError_t cerr; 
 
    int block_size = 16; 
    dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size ); 
    dim3 threads( block_size, block_size ); 
    kernel_zvetseni1<<< blocks, threads >>>( colorPic, bwPic ); 
 
    if ( ( cerr = hipGetLastError() ) != hipSuccess ) 
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) ); 
 
    hipDeviceSynchronize(); 
} 
 
void cu_run_zacerneni( CUDA_Pic colorPic) 
{ 
    hipError_t cerr; 
 
    int block_size = 16; 
    dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size ); 
    dim3 threads( block_size, block_size ); 
    kernel_zacerneni<<< blocks, threads >>>( colorPic ); 
 
    if ( ( cerr = hipGetLastError() ) != hipSuccess ) 
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) ); 
 
    hipDeviceSynchronize(); 
} 
 
 
void cu_run_zvetseni2( CUDA_Pic colorPic, CUDA_Pic bwPic1 ) 
{ 
    hipError_t cerr; 
 
    int block_size = 16; 
    dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size ); 
    dim3 threads( block_size, block_size ); 
    kernel_zvetseni2<<< blocks, threads >>>( colorPic, bwPic1 ); 
 
    if ( ( cerr = hipGetLastError() ) != hipSuccess ) 
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) ); 
 
    hipDeviceSynchronize(); 
}
