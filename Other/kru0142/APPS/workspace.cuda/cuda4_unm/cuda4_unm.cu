#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include "pic_type.h"
#include "animation.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale( CUDA_Pic colorPic, CUDA_Pic bwPic )
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= colorPic.Size.y ) return;
	if ( x >= colorPic.Size.x ) return;

	// Get point from color picture
	//uchar3 bgr = colorPic.P_uchar3[ y * colorPic.Size.x + x ];
	uchar3 bgr = colorPic.at3(y,x);

	// Store BW point to new image
	//bwPic.P_uchar1[ y * bwPic.Size.x + x ].x = bgr.x * 0.11 + bgr.y * 0.59 + bgr.z * 0.30;
if( y <= x*colorPic.Size.y/colorPic.Size.x)
{
	bwPic.at3(y,x).x = bgr.x;
	bwPic.at3(y,x).y = bgr.y;
	bwPic.at3(y,x).z = bgr.z;

}
else
{
	bwPic.at3(y,x).x = 	bwPic.at3(y,x).y = 	bwPic.at3(y,x).z = bgr.x * 0.11 + bgr.y * 0.59 + bgr.z * 0.30;


}






	//bwPic.at1(y,x).x = bgr.x * 0.11 + bgr.y * 0.59 + bgr.z * 0.30;




}








__global__ void kernel_rotate( CUDA_Pic original, CUDA_Pic rotated,int rotnum)
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= original.Size.y ) return;
	if ( x >= original.Size.x ) return;
	if ( y >= rotated.Size.x ) return;
	if ( x >= rotated.Size.y ) return;

	// Get point from color picture
	//uchar3 bgr = colorPic.P_uchar3[ y * colorPic.Size.x + x ];
	uchar3 bgr = original.at3(y,x);

	// Store BW point to new image
	//bwPic.P_uchar1[ y * bwPic.Size.x + x ].x = bgr.x * 0.11 + bgr.y * 0.59 + bgr.z * 0.30;

	if(rotnum ==1 )
	{
		rotated.at3(original.Size.x - x,y).x = bgr.x;
		rotated.at3(original.Size.x - x,y).y = bgr.y;
		rotated.at3(original.Size.x - x,y).z = bgr.z;

	}
	else if(rotnum ==3)
	{
		rotated.at3(x,original.Size.y-y).x = bgr.x;
		rotated.at3(x,original.Size.y-y).y = bgr.y;
		rotated.at3(x,original.Size.y-y).z = bgr.z;

	}










	//bwPic.at1(y,x).x = bgr.x * 0.11 + bgr.y * 0.59 + bgr.z * 0.30;




}

__global__ void kernel_shrink( CUDA_Pic original, CUDA_Pic shrunk,int widthheight)
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= original.Size.y ) return;
	if ( x >= original.Size.x ) return;

	// Get point from color picture
	//uchar3 bgr = colorPic.P_uchar3[ y * colorPic.Size.x + x ];


	// Store BW point to new image
	//bwPic.P_uchar1[ y * bwPic.Size.x + x ].x = bgr.x * 0.11 + bgr.y * 0.59 + bgr.z * 0.30;

	//printf("WIDTHHEIGHT = %d\n",widthheight);
	//printf("Taking TOP x:%d,y:%d into X:%d,Y%d\n",x,y*2-1,x,y);
	//printf("Taking BOT x:%d,y:%d into X:%d,Y%d\n",x,y*2,x,y);


	if(widthheight ==1 )
	{
		uchar3 bgrl = original.at3(y,x*2);
		uchar3 bgrr = original.at3(y,x*2+1);

		shrunk.at3(y,x).x = (bgrl.x+bgrr.x)/2;
		shrunk.at3(y,x).y = (bgrl.y+bgrr.y)/2;
		shrunk.at3(y,x).z = (bgrl.z+bgrr.z)/2;

	}
	else if(widthheight ==2 )
	{
		uchar3 bgrt = original.at3(y*2,x);
		uchar3 bgrb = original.at3(y*2+1,x);

		shrunk.at3(y,x).x = (bgrt.x+bgrb.x)/2;
		shrunk.at3(y,x).y = (bgrt.y+bgrb.y)/2;
		shrunk.at3(y,x).z = (bgrt.z+bgrb.z)/2;

	}
	else if(widthheight ==3)
		{
			uchar3 bgrtl = original.at3(y*2,x*2);
			uchar3 bgrbr = original.at3(y*2+1,x*2+1);
			uchar3 bgrbl = original.at3(y*2+1,x*2);
			uchar3 bgrtr = original.at3(y*2,x*2+1);

			shrunk.at3(y,x).x = bgrtl.x;//(bgrtl.x+bgrbl.x +bgrtr.x+bgrbr.x)/4;
			shrunk.at3(y,x).y = bgrtl.y;//(bgrtl.y+bgrbl.y+bgrtr.y+bgrbr.y)/4;
			shrunk.at3(y,x).z = bgrtl.z;//(bgrtl.z+bgrbl.z+bgrtr.z+bgrbr.z)/4;

		}









	//bwPic.at1(y,x).x = bgr.x * 0.11 + bgr.y * 0.59 + bgr.z * 0.30;




}






void cu_run_grayscale( CUDA_Pic colorPic, CUDA_Pic bwPic )
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	int block_size = 16;
	dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );
	kernel_grayscale<<< blocks, threads >>>( colorPic, bwPic );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();
}
void cu_run_rotate( CUDA_Pic original, CUDA_Pic rotated,int rotnum)
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	int block_size = 16;
	dim3 blocks( ( original.Size.x + block_size - 1 ) / block_size, ( original.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );

		kernel_rotate<<< blocks, threads >>>( original, rotated ,rotnum);




	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();
}
void cu_run_shrink( CUDA_Pic original, CUDA_Pic shrunk,int widthheight)
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	int block_size = 16;
	dim3 blocks( ( shrunk.Size.x + block_size - 1 ) / block_size, ( shrunk.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );

		kernel_shrink<<< blocks, threads >>>( original, shrunk ,widthheight);




	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();
}






__global__  void kernel_insertimage( CUDA_Pic bigPic, CUDA_Pic smallPic, int2 position )
{
    
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= smallPic.Size.y ) { return;}
	if ( x >= smallPic.Size.x ) { return;}
	int by = y + position.y;
	int bx = x + position.x;
	if ( by >= bigPic.Size.y || by < 0 ) { return;}
	if ( bx >= bigPic.Size.x || bx < 0 ) { return;}

	// Get point from small image
	uchar4 fg_bgra = smallPic.P_uchar4[ y * smallPic.Size.x + x ];
   // printf("Uchar4 x = %d,y = %d,z = %d,w = %d",fg_bgra.x,fg_bgra.y,fg_bgra.z,fg_bgra.w);	
    uchar3 bg_bgr = bigPic.P_uchar3[ by * bigPic.Size.x + bx ];
   // printf("Uchar3 x = %d,y = %d,z = %d",bg_bgr.x,bg_bgr.y,bg_bgr.z);
	uchar3 bgr = { 0, 0, 0 };

	// compose point from small and big image according alpha channel
	bgr.x = fg_bgra.x * fg_bgra.w / 255 + bg_bgr.x * ( 255 - fg_bgra.w ) / 255;
	bgr.y = fg_bgra.y * fg_bgra.w / 255 + bg_bgr.y * ( 255 - fg_bgra.w ) / 255;
	bgr.z = fg_bgra.z * fg_bgra.w / 255 + bg_bgr.z * ( 255 - fg_bgra.w ) / 255;

	// Store point into image
	bigPic.P_uchar3[ by * bigPic.Size.x + bx ] = bgr;
}

void cu_insertimage( CUDA_Pic bigPic, CUDA_Pic smallPic, int2 position )
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	int block_size = 32;
	dim3 blocks( ( smallPic.Size.x + block_size - 1 ) / block_size, ( smallPic.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );
	kernel_insertimage<<< blocks, threads >>>( bigPic, smallPic, position );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();
}







__global__ void kernel_rotaround( CUDA_Pic colorPic, CUDA_Pic bwPic,  float degr, int2 position ) 
{ 
     // X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= colorPic.Size.y ) { return;}
	if ( x >= colorPic.Size.x ) { return;}



 float deg = degr*3.14/180; 
  
 
    int xc =colorPic.Size.x/2; 
    int yc =colorPic.Size.y/2; 
 
    int nx = ((float)x-xc)*cos(deg) - ((float)y-yc)*sin(deg) + xc*2+position.x; 
    int ny = ((float)x-xc)*sin(deg) + ((float)y-yc)*cos(deg) + yc*2+position.y; 

	// Get point from small image
	uchar4 fg_bgra = colorPic.at4(y,x);
   // printf("Uchar4 x = %d,y = %d,z = %d,w = %d",fg_bgra.x,fg_bgra.y,fg_bgra.z,fg_bgra.w);	
    uchar3 bg_bgr = bwPic.at3(ny,ny);
   // printf("Uchar3 x = %d,y = %d,z = %d",bg_bgr.x,bg_bgr.y,bg_bgr.z);
	uchar3 bgr = { 0, 0, 0 };

	// compose point from small and big image according alpha channel
    bgr.x = fg_bgra.x * fg_bgra.w / 255 + bg_bgr.x * ( 255 - fg_bgra.w ) / 255;
	bgr.y = fg_bgra.y * fg_bgra.w / 255 + bg_bgr.y * ( 255 - fg_bgra.w ) / 255;
	bgr.z = fg_bgra.z * fg_bgra.w / 255 + bg_bgr.z * ( 255 - fg_bgra.w ) / 255;
   
if (nx >= 0 && nx < bwPic.Size.x && ny >= 0 && ny < bwPic.Size.y) 
{
	
    bwPic.P_uchar3[ ny * bwPic.Size.x + nx ] = bgr;
}
	
} 


void cu_run_rotaround( CUDA_Pic colorPic, CUDA_Pic bwPic, float degr, int2 position ) 
{ 
    hipError_t cerr; 
 
    int block_size = 16; 
    dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size ); 
    dim3 threads( block_size, block_size ); 
    kernel_rotaround<<< blocks, threads >>>( colorPic, bwPic, degr ,position); 
 
    if ( ( cerr = hipGetLastError() ) != hipSuccess ) 
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) ); 
 
    hipDeviceSynchronize(); 
} 











